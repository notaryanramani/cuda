#include <iostream>
#include <hip/hip_runtime.h>

int main(){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << prop.maxThreadsPerBlock << std::endl;
    std::cout << prop.maxGridSize[0] << std::endl;
}