#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void addVectors(int *a, int *b, int *c, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        c[idx] = a[idx] + b[idx];
    }
    
}

int main(){
    int n = 1000000;
    size_t size = n * sizeof(int);

    int *ha = (int *)malloc(size);
    int *hb = (int *)malloc(size);
    int *hc = (int *)malloc(size);

    for (int i=0; i<n; i++){
        ha[i] = i;
        hb[i] = i * 2;
    }

    int *da, *db, *dc;
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipError_t err =  hipMalloc(&dc, size);
    if (err != hipSuccess){
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    int blockPerGrid = (n + threadPerBlock - 1) / threadPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);

    hipEventRecord(start);

    // Kernel call
    addVectors<<<blockPerGrid, threadPerBlock>>>(da, db, dc, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "GPU Elapsed time: " << elapsedTime << " ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    auto start_c = std::chrono::high_resolution_clock::now();
    int a;
    for (int i = 0; i < n; i++) {
        a = ha[i] + hb[i];
    }
    auto end_c = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_c - start_c;
    std::cout << "CPU execution time: " << duration.count() * 1000 << " ms" << std::endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(ha);
    free(hb);
    free(hc);

    return 0;
}